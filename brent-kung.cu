#include "hip/hip_runtime.h"

/*
brent-kung.cu

Jordan Kremer
Dalton Bohning

Usage:
    Flags:
        -DARRAY_SIZE
        -SECTION_SIZE 

    Ex:
        nvcc -DARRAY_SIZE=2000 -DSECTION_SIZE=2048 -o brent-kung brent-kung.cu

    Note:
        Section size should not exceed 2048 
*/


#include <hip/hip_runtime.h>
#include <stdio.h>


//#define SECTION_SIZE 100
//#define ARRAY_SIZE 100


#define handleError(CUDA_FUNCTION) {\
    hipError_t THE_ERROR = (hipError_t) CUDA_FUNCTION;\
    if (THE_ERROR != hipSuccess) \
    {\
        printf("%s in %s at line %d\n", hipGetErrorString(THE_ERROR),__FILE__,__LINE__);\
        exit(EXIT_FAILURE);\
    }\
}



//Credit: https://github.com/aramadia/udacity-cs344/blob/master/Unit2%20Code%20Snippets/gputimer.h
struct GpuTimer
{
      hipEvent_t start;
      hipEvent_t stop;
 
      GpuTimer()
      {
            hipEventCreate(&start);
            hipEventCreate(&stop);
      }
 
      ~GpuTimer()
      {
            hipEventDestroy(start);
            hipEventDestroy(stop);
      }
 
      void Start()
      {
            hipEventRecord(start, 0);
      }
 
      void Stop()
      {
            hipEventRecord(stop, 0);
      }
 
      float Elapsed()
      {
            float elapsed;
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed, start, stop);
            return elapsed;
      }
};

GpuTimer timer_kernelExecution;
GpuTimer timer_kernelTotal;


//An iterative version of parallel scan addition
__host__
void sequential_scan(float *X, float *Y){
  int i = 1, acc = Y[0] = X[0];
  
  while (i < ARRAY_SIZE) 
    Y[i++] = acc = acc + X[i];
}

//Runs the iterative version and verifies the results
__host__
bool verify(float *X, float *Y){
  float *Y_ = (float*) malloc(ARRAY_SIZE * sizeof(float));
  sequential_scan(X, Y_);
  for (int i = 0; i < ARRAY_SIZE; ++i){
    if (Y[i] != Y_[i]) {
      printf("Expected %.0f but got %.0f at Y[%d]\n", Y_[i], Y[i], i);
      return false;
    }
  }
  free(Y_);
  return true;
}

__global__ 
void Brent_Kung_scan_kernel(float *X, float *Y)
{
    __shared__ float XY[SECTION_SIZE];

    int i = 2*blockIdx.x*blockDim.x + threadIdx.x;
    if(i < ARRAY_SIZE)
    {
        XY[threadIdx.x] = X[i];
    }

    if(i + blockDim.x < ARRAY_SIZE)
    {
        XY[threadIdx.x+blockDim.x] = X[i + blockDim.x];
    }

    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        __syncthreads();
        int index = (threadIdx.x+1) * 2 * stride - 1;
        if(index < SECTION_SIZE)
        {
            XY[index] += XY[index - stride];
        }
    }

    for(int stride = SECTION_SIZE/4; stride > 0; stride /= 2)
    {
        __syncthreads();
        int index = (threadIdx.x+1) * stride * 2 - 1;
        if(index + stride < SECTION_SIZE) 
        {
            XY[index + stride] += XY[index];
        }
    }

    __syncthreads();
    if(i < ARRAY_SIZE)
    {
        Y[i] = XY[threadIdx.x];
    }
    
    if(i + blockDim.x < ARRAY_SIZE)
    {
        Y[i + blockDim.x] = XY[threadIdx.x + blockDim.x];
    }

}


void inclusive_scan(float *host_X, float *host_Y)
{
    float *X, *Y;
    int mallocSize = ARRAY_SIZE * sizeof(float);

    timer_kernelTotal.Start();

    handleError(hipMalloc((void **)&X, mallocSize));
    handleError(hipMalloc((void **)&Y, mallocSize));

    handleError(hipMemcpy(X, host_X, mallocSize, hipMemcpyHostToDevice));
   
    //Book says SECTION_SIZE/2 OK, but not sure about
    //other dimensions and blocks per grid
    dim3 threadsPerBlock(SECTION_SIZE/2, 1, 1);
    dim3 blocksPerGrid(100,1,1);

    timer_kernelExecution.Start();
    Brent_Kung_scan_kernel<<<blocksPerGrid, threadsPerBlock>>>(X, Y);
    timer_kernelExecution.Stop();

    handleError(hipMemcpy(host_Y, Y, mallocSize, hipMemcpyDeviceToHost));
    handleError(hipFree(X));
    handleError(hipFree(Y));

    timer_kernelTotal.Stop();
}

void printArray(float *A){
  for(int i = 0; i < ARRAY_SIZE; ++i) {
    printf("%.0f ", A[i]);
    if((i+1) % 10 == 0){
      printf("\n");
    }
  }
  printf("\n");
}

int main(void)
{
    float *host_X = (float*) malloc(ARRAY_SIZE * sizeof(float));
    float *host_Y = (float*) malloc(ARRAY_SIZE * sizeof(float));

    for(int i = 0; i < ARRAY_SIZE; ++i)
    {
        host_X[i] = i + i %4; //change
    }


    inclusive_scan(host_X, host_Y);

    //Make sure the results are correct
    if (1) {
      printArray(host_Y);
      if (verify(host_X, host_Y))
        printf("ALL CORRECT!\n");
      else
        printf("FAIL!\n");
    }

    float kernelExec = timer_kernelExecution.Elapsed();
    float kernelTotal = timer_kernelTotal.Elapsed();
    float kernelMem = kernelTotal - kernelExec;

    printf("Kernel Execution (ms): %f\n", kernelExec);
    printf("Kernel Memory (ms):    %f\n", kernelMem);
    printf("Kernel Total (ms):     %f\n", kernelTotal);

    free(host_X);
    free(host_Y);
}
